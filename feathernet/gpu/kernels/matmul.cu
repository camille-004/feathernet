
#include <hip/hip_runtime.h>
__global__ void matmul(float* A, float* B, float* C, int A_rows, int A_cols, int B_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols) {
        float sum = 0.0;
        for (int k = 0; k < A_cols; k++) {
            sum += A[row * A_cols + k] * B[k * B_cols + col];
        }
        C[row * B_cols + col] = sum;
    }
}
